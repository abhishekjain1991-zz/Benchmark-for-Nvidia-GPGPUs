#include "hip/hip_runtime.h"
#include<iostream>

#include <stdio.h>
#include <gl/glut.h>
#include <math.h>
#include<vector>
#include <hip/hip_runtime_api.h>
#include<windows.h>
#include<time.h>

class Complex1 
{
public:
    float   r;
    float   i;
    __device__ Complex1( float a, float b ) : r(a), i(b)  {}
    __device__ Complex1(const Complex1& x) : r(x.r), i(x.i) {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ Complex1 operator*(const Complex1& a) {
        return Complex1(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ Complex1 operator+(const Complex1& a) {
        return Complex1(r+a.r, i+a.i);
    }
};
using namespace std;
int StartX = -1;
int StartY = -1;
int EndX = -1;
int EndY = -1;
clock_t t,t1;
int nx, ny,c=0,r,g=256,b=256,noofani=0,updateRate=200;
double *arrx,*arry,*arrop;
//GLdouble realMax=0.75f,realMin=-2.25f,imagMax=1.25f,imagMin=-1.25f,realInc,imagInc;
GLdouble realMax=1.0f,realMin=-2.0f,imagMax=1.8f,imagMin=-1.2f,realInc,imagInc;
void timer(int)
{noofani++;
if(noofani<5)
{
  // Adjust rotation angles as needed here
  // Then tell glut to redisplay
  glutPostRedisplay();
  // And reset tht timer
  glutTimerFunc(1000.0 / updateRate, timer, 0);
}
else if(noofani==5)
{//cout << "done benchmarking " << (tp.tv_sec+tp.tv_usec/1000000.0) - startSec << " seconds" << endl;
DWORD points=GetTickCount();
t1=clock();
cout << "done benchmarking and points afer "<<noofani <<" iterations are "<< t1-t<< endl;
}

else
	return;
}
class memory
{public:
double minx,miny,maxx,maxy;
memory(double a, double b, double c, double d):minx(a),miny(b),maxx(c),maxy(d)
{}

};
vector<memory> m1;

__global__ void kernal(double *dev_arrx,double *dev_arry, double *dev_arrop)
{

			int tid = threadIdx.x + blockIdx.x * blockDim.x; 
			
			int i = tid / 512;
			int j = tid % 512;

			Complex1 c(dev_arrx[j],dev_arry[i]),op(0,0),temp(0,0);
		    int cnt=0;
			
			 while(((op.r)*(op.r))+((op.i)*(op.i))<=4 && cnt<=2000)
			 {
				 op=(temp*temp)+c;
				 temp=op;
				 cnt++;
			 }
			 dev_arrop[tid]=cnt;
	
}


// Called to draw scene

void RenderScene(void) {
    

  
    // Clear the window with current clearing color
    glClear(GL_COLOR_BUFFER_BIT);
    
	int arropcnt=0,cnt=0;


    realInc = (realMax - realMin) / (GLdouble)nx;
    imagInc = (imagMax - imagMin) / (GLdouble)ny;
    // Call only once for all remaining points
    
    for(int i=0;i<ny;i++)
	{
	arry[i]=imagMin+(i*imagInc);
	}
	for(int j=0;j<nx;j++)
	{
			arrx[j]=realMin + (j*realInc);
			
	}


	double *dev_a, *dev_b, *dev_c;
	hipMalloc( (void**)&dev_a, 512*sizeof(double) );
	hipMalloc( (void**)&dev_b, 512*sizeof(double) );
	hipMalloc( (void**)&dev_c, 512*512*sizeof(double) );

	hipMemcpy( dev_a, arrx, 512 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy( dev_b, arry, 512 * sizeof(double), hipMemcpyHostToDevice);

	kernal<<<8192,32>>>(dev_a, dev_b, dev_c);

	hipMemcpy (arrop, dev_c, 512 * 512 * sizeof(double), hipMemcpyDeviceToHost );


	
	arropcnt=0;
	glBegin(GL_POINTS);
	for(int i=0;i<ny;i++)
	{
		for( int j=0;j<nx;j++,arropcnt++)
		{

			if(arrop[arropcnt]<2000 && arrop[arropcnt]>0)
			{r=arrop[arropcnt];
			r=r%16;
			switch (r)
				{case 1: glColor3f(25, 0, 26);
						break;
				case 2: glColor3f(9, 0, 47);
						break;

				case 3: glColor3f(4, 0, 73);
						break;
				case 4: glColor3f(0, 7, 100);
						break;
				case 5: glColor3f(0, 44, 138);
						break;
				case 6: glColor3f(0, 82, 177);
						break;

				case 7: glColor3f(0, 125, 209);
						break;
				case 8: glColor3f(0, 181, 229);
						break;
				case 9: glColor3f(0, 236, 248);
						break;
				case 10: glColor3f(241, 233, 0);
						break;

				case 11: glColor3f(248, 201, 0);
						break;
				case 12: glColor3f(255, 170, 0);
						break;
				case 13: glColor3f(204, 128, 0);
						break;
				case 14: glColor3f(153, 87, 0);
						break;

				case 15: glColor3f(106, 52, 0);
						break;
				case 0: glColor3f(66, 30, 0);
						break;
				default: glColor3f(66, 30, 0);break;
			}

				
				
				
			}
			else
				glColor3f(0,0,0);

			 glVertex2d( j, i );
		}
	}

   
    // Done drawing points
    glEnd();

    glMatrixMode( GL_PROJECTION );
    glLoadIdentity();
    glOrtho( 0, nx, ny, 0, -nx, nx );

    glMatrixMode( GL_MODELVIEW );
   glLoadIdentity();
	//----selection
    if( StartX > 0 && StartY > 0 && EndX > 0 && EndY > 0 )
    {
        glLogicOp(GL_XOR);
        glEnable(GL_COLOR_LOGIC_OP);
        glColor3f(1.0, 1.0, 1.0);
        glLineWidth(1.0);
        glBegin(GL_LINE_LOOP);
		
        glVertex2i(StartX, StartY);
        glVertex2i(EndX, StartY);
        glVertex2i(EndX, EndY);
        glVertex2i(StartX, EndY);
		
        glEnd();
        glDisable(GL_COLOR_LOGIC_OP);
    }

    // Flush drawing commands
    glutSwapBuffers();
	realMax=realMax/1.8+0.25;
	imagMax=imagMax/1.8+0.25;
	realMin=realMin/1.8-0.25;
	imagMin=imagMin/1.8-0.25;
}



void mouse( int button, int state, int x, int y )
{
    if( button == GLUT_LEFT && state == GLUT_DOWN )
    {
        StartX = x;
        StartY = y;
		
	
    }
    if( button == GLUT_LEFT && state == GLUT_UP )
    {
	if(StartX<EndX)
	{realMin=arrx[StartX];
	realMax=arrx[EndX];
	}
	else
	{realMin=arrx[EndX];
	realMax=arrx[StartX];
	}
	if(StartY<EndY)
	{
	imagMin=arry[StartY];
	imagMax=arry[EndY];
	}
	else
	{imagMin=arry[EndY];
	imagMax=arry[StartY];
	}

	m1.push_back(memory(realMin,imagMin,realMax,imagMax));
        StartX = -1;
        StartY = -1;
        EndX = -1;
        EndY = -1;
		glutPostRedisplay();
    }
}

void motion( int x, int y )
{
    EndX = x;
	cout<<StartX<<" "<<StartY<<" ";
	if(StartY-y>0 && StartX-EndX>0)
    EndY = StartY-(StartX-EndX);
	else if(StartY-y>0 && StartX-EndX<0)
    EndY = StartY+(StartX-EndX);
	else if(StartY-y<0 && StartX-EndX>0)
	EndY = StartY+(StartX-EndX);
	else
	EndY = StartY-(StartX-EndX);

	
	glutPostRedisplay();
	cout<<EndX<<" "<<EndY<<endl;
}



void Key(unsigned char key, int x, int y) {
	

	if(key=='b'|| key== 'B')
	{
	if(m1.size()>1)
		{imagMin=m1[m1.size()-2].miny;
		realMin=m1[m1.size()-2].minx;
		imagMax=m1[m1.size()-2].maxy;
		realMax=m1[m1.size()-2].maxx;
	}
		if(m1.size()!=1)
		m1.erase(m1.begin()+(m1.size()-1));
		
	}
	if(key=='q'|| key== 'Q')
		exit(0);


	 glutPostRedisplay();
    
}


void ChangeSize(int w, int h) {
    nx = 512;
    ny = 512;
  
	if(arrx!=0)
	{delete[] arrx;}
	if(arry!=0)
		delete[] arry;
	if(arrop!=0)
		delete[] arrop;
	
    arrx=new double [512];
	arry=new double [512];
	arrop=new double [512*512];
	if(m1.size()>0)
		m1.clear();
	
	m1.push_back(memory(realMin,imagMin,realMax,imagMax));
 
    // Set Viewport to window dimensions
    glViewport(0, 0, w, h);

    // Reset projection matrix stack
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    // Establish clipping volume (left, right, bottom, top, near, far)
    
        glOrtho(0,w,0,h,0,w);
    
    
        
    // Reset Model view matrix stack
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

int main(int argc, char* argv[]) {
    
	t=clock();
	glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(512, 512);
	glutCreateWindow("CUDA LAB");
    glutReshapeFunc(ChangeSize);
	glutKeyboardFunc(Key);
	glutMouseFunc( mouse );
    glutMotionFunc( motion );
    glutDisplayFunc(RenderScene);
	glutTimerFunc(1000.0 / updateRate, timer, 0);
    glutMainLoop();

    return 0;
}
